#include "hip/hip_runtime.h"

#include <iostream>

#include "file1.h"
#include "file2.h"

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#define IMPORT __declspec(dllimport)
#else
#define EXPORT
#define IMPORT
#endif

result_type __device__ file1_func(int x);
result_type_dynamic __device__ file2_func(int x);

IMPORT void __host__ cuda_dynamic_lib_func();

static __global__ void mixed_kernel(result_type& r, int x)
{
  r = file1_func(x);
  result_type_dynamic rd = file2_func(x);
}

EXPORT int mixed_launch_kernel(int x)
{
  cuda_dynamic_lib_func();

  result_type r;
  mixed_kernel<<<1, 1>>>(r, x);
  return r.sum;
}
