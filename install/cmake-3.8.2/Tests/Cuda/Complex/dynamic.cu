
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

int dynamic_base_func(int);

EXPORT int __host__ cuda_dynamic_host_func(int x)
{
  return dynamic_base_func(x);
}

static __global__ void DetermineIfValidCudaDevice()
{
}

EXPORT void cuda_dynamic_lib_func()
{
  DetermineIfValidCudaDevice<<<1, 1>>>();
  hipError_t err = hipGetLastError();
  if (err == hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
  }
}
